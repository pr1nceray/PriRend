#include "hip/hip_runtime.h"
#include "./Materials.cuh"
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

std::unordered_map<std::string, TextInfo *> Material::currentMaterials;

const std::unordered_map<std::string, TextInfo *> & Material::getTextures() {
    return currentMaterials;
}
void Material::loadDiffuse(const std::string & fileName) {
    Diffuse = checkInScene(fileName);
}

void Material::loadNormal(const std::string & fileName) {
    Normal = checkInScene(fileName);
}

void Material::loadSpecular(const std::string & fileName) {
    Specular = checkInScene(fileName);
}

TextInfo Material::loadImage(const std::string & fileName) {
    int width, height, numChannel;
    uint8_t * imageData = stbi_load(std::string("./assets/Textures/" + fileName).c_str(), & width, &height, &numChannel, 3);
    if (imageData == NULL) {
        throw std::runtime_error("Error loading Texture file " + fileName  + ". See logs for more.");
    }
    float * newImageData = new float[width * height * 3];
    convert(imageData, width * height * 3, newImageData);
    stbi_image_free(imageData);
    return TextInfo{newImageData, width, height};
}

TextInfo *Material::checkInScene(const std::string & fileName) {
    if (Material::currentMaterials.find(fileName) != Material::currentMaterials.end()) {
        return Material::currentMaterials.find(fileName)->second;
    }
    TextInfo *texture = new TextInfo();
    *texture = loadImage(fileName);
    Material::currentMaterials[fileName] = texture;
    return texture;
}

// will cause slowdown when loading many images.
// consider speeding up with cuda kernel?
void Material::convert(uint8_t * source, size_t max, float * out) {
    for(size_t i = 0; i < max; ++i) {
        out[i] = source[i]/255.0f;
    }
}

const TextInfo * Material::getDiffuse() const {
    return Diffuse;
}