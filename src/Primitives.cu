#include "hip/hip_runtime.h"
#include "Primitives.cuh"

 __host__ __device__ void normalizeRayDir(Ray & ray) {
    ray.Dir = glm::normalize(ray.Dir);
}

/*
* Flip a ray along the normal for PURE reflective materials. 
*/
glm::vec3 flipRayNormal(const Ray & ray, const glm::vec3 & normal) {
    return ray.Dir - (2.0f * (glm::dot(ray.Dir, normal) * normal));
}

__device__ const bool isZero(const glm::vec3 * in) {
    return ((fabs(in->x) < epsil) && (fabs(in->y) < epsil) && (fabs(in->z) < epsil)); 
}

__host__ uint8_t generateRandomNumH() {
    return static_cast<uint8_t>(rand() % 255);
}

__host__ float generateRandomFloatH() {
    return (static_cast<float>(std::rand() - RAND_MAX/2)/RAND_MAX);
}
__device__ float generateRandomFloatD(hiprandState * const state) {
    return hiprand_uniform(state) - .500000000001f;
}   

__device__ float generateNormalFloatD(hiprandState * const state) {
    return hiprand_normal(state);
}   

/*
* The point of this is to get a y value from a Z score, with
* the higher the z score, the lower the return value
* if fabs( rand value) < epsil, return 1/epsil to avoid any rounding errors
* returning 100000f isnt a great solution, but it will do
*/
__device__ float generateInvNormalFloatD(hiprandState * const state) {
    float rand = generateNormalFloatD(state);
    return fabs(rand)<epsil?1000000.0f:1.0f/rand;
}  



__host__ glm::vec3 generateRandomVecH() {
    return glm::normalize(glm::vec3(
        generateRandomFloatH(), generateRandomFloatH(), generateRandomFloatH()));
}
__device__ glm::vec3 generateRandomVecD(hiprandState * const state) {
    return glm::normalize(glm::vec3(
        generateRandomFloatD(state), generateRandomFloatD(state), generateRandomFloatD(state)));
}

__device__ glm::vec3 generateNormalVecD(hiprandState * state) {
    return glm::normalize(glm::vec3(
        generateNormalFloatD(state), generateNormalFloatD(state), generateNormalFloatD(state)));
}

__device__ glm::vec3 generateInvNormalVecD(hiprandState * state) {
    return glm::normalize(glm::vec3(
        generateInvNormalFloatD(state), generateInvNormalFloatD(state), generateInvNormalFloatD(state)));
}